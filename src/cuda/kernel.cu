#include "hip/hip_runtime.h"

// Copyright (c) 2024 The Pastel developers
// Distributed under the MIT software license, see the accompanying
// file COPYING or https://www.opensource.org/licenses/mit-license.php.
#include <cstdint>
#include <vector>
#include <string>
#include <bitset>
#include <iostream>

#include <hip/hip_runtime.h>
#include <>

#include <src/cuda/memutils.h>
#include <src/cuda/blake2b_device.h>
#include <src/equihash/equihash.h>

using namespace std;

// Get the number of available CUDA devices
int getNumCudaDevices()
{
    int numDevices = 0;
    hipGetDeviceCount(&numDevices);
    return numDevices;
}

// Get the maximum number of threads per block supported by the device
int getMaxThreadsPerBlock(int deviceId)
{
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, deviceId);
    return deviceProp.maxThreadsPerBlock;
}

template<typename EquihashType>
bool EhDevice<EquihashType>::allocate_memory()
{
    try
    {
        // Allocate device memory for blake2b state
        initialState = make_cuda_unique<blake2b_state>(1);
        // Allocate device memory for initial hash values
        hashes = make_cuda_unique<uint32_t>(EquihashType::NHashWords);
        // Allocate device memory for XORed hash values
        xoredHashes = make_cuda_unique<uint32_t>(EquihashType::NHashWords);

        const uint32_t maxCollisionsPerBucket = 10000;

        vCollisionPairs.resize(EquihashType::NBucketCount);
        for (uint32_t i = 0; i < EquihashType::NBucketCount; ++i)
            vCollisionPairs[i] = make_cuda_unique<uint32_t>(maxCollisionsPerBucket);

        vCollisionCounters.resize(EquihashType::WK, vector<unique_ptr<uint32_t, CudaDeleter>>(EquihashType::NBucketCount));
        for (uint32_t round = 0; round < EquihashType::WK; ++round)
        {
            for (uint32_t i = 0; i < EquihashType::NBucketCount; ++i)
                vCollisionCounters[round][i] = make_cuda_unique<uint32_t>(1);
        }

        // Allocate device memory for solutions and solution count
        solutions = make_cuda_unique<typename EquihashType::solution>(MAXSOLUTIONS);
        solutionCount = make_cuda_unique<uint32_t>(1);

        return true;
    }
    catch (const std::exception& e)
    {
        std::cerr << "Memory allocation failed: " << e.what() << std::endl;
        return false;
    }
}

// Calculate the grid and block dimensions based on the problem size and device capabilities
void calculateGridAndBlockDims(dim3& gridDim, dim3& blockDim, size_t nHashes, int nThreadsPerHash, int deviceId)
{
    int nMaxThreadsPerBlock = getMaxThreadsPerBlock(deviceId);
    int nBlocks = (nHashes + nThreadsPerHash - 1) / nThreadsPerHash;

    blockDim.x = min(nThreadsPerHash, nMaxThreadsPerBlock);
    gridDim.x = (nBlocks + blockDim.x - 1) / blockDim.x;
}

// CUDA kernel to generate initial hashes from blake2b state
template<typename EquihashType>
__global__ void cudaKernel_generateInitialHashes(const blake2b_state* state, uint32_t* hashes)
{
    const uint32_t hashIdx = threadIdx.x + blockIdx.x * blockDim.x;

    if (hashIdx >= EquihashType::NHashes)
        return;

    const uint32_t blockIndex = hashIdx / EquihashType::IndicesPerHashOutput;

    blake2b_state localState = *state;
    blake2b_update_device(&localState, reinterpret_cast<const uint8_t*>(&blockIndex), sizeof(blockIndex));

    uint8_t hash[EquihashType::HashOutput];  
    blake2b_final_device(&localState, hash, EquihashType::HashOutput);

    const uint32_t outputIdx = hashIdx * EquihashType::HashWords;
    for (uint32_t i = 0; i < EquihashType::HashWords; ++i)
        hashes[outputIdx + i] = (reinterpret_cast<uint32_t*>(hash))[i];
}

template<typename EquihashType>
void EhDevice<EquihashType>::generateInitialHashes()
{
    const uint32_t numThreads = (EquihashType::NHashes + threadsPerBlock - 1) / threadsPerBlock * threadsPerBlock;

    dim3 gridDim((numThreads + threadsPerBlock - 1) / threadsPerBlock);
    dim3 blockDim(threadsPerBlock);

    cudaKernel_generateInitialHashes<EquihashType><<<gridDim, blockDim>>>(initialState.get(), hashes.get());
}

template <typename EquihashType>
__global__ void cudaKernel_detectCollisions(
    const uint32_t* hashes, uint32_t* collisionPairs, uint32_t* collisionCounts,
    const uint32_t startIdx, const uint32_t endIdx, const uint32_t collisionBitMask)
{
    const uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    const uint32_t idx = startIdx + tid;

    if (idx >= endIdx)
        return;

    const uint32_t hash = hashes[idx];
    const uint32_t maskedHash = hash & collisionBitMask;

    const uint32_t bucketIdx = idx / EquihashType::NBucketSize;
    const uint32_t bucketOffset = bucketIdx * EquihashType::NBucketSize;

    for (uint32_t i = bucketOffset; i < idx; ++i)
    {
        const uint32_t otherHash = hashes[i];
        const uint32_t otherMaskedHash = otherHash & collisionBitMask;

        if (maskedHash == otherMaskedHash)
        {
            const uint32_t collisionIdx = atomicAdd(collisionCounts, 1);
            collisionPairs[collisionIdx] = (idx << 16) | i;
        }
    }    
}

template <typename EquihashType>
void EhDevice<EquihashType>::detectCollisions()
{
    if (m_vCollisionPairsOffsets.size() <= round)
        m_vCollisionPairsOffsets.resize(round + 1, 0);
    // Calculate the accumulated collision pair offset for the current round
    uint32_t collisionPairsOffset = 0;
    if (round > 0)
        collisionPairsOffset = m_vCollisionPairsOffsets[round - 1];

    for (uint32_t bucketIdx = 0; bucketIdx < EquihashType::NBucketCount; ++bucketIdx)
    {
        const uint32_t startIdx = bucketIdx * EquihashType::NBucketSize;
        const uint32_t endIdx = min(startIdx + EquihashType::NBucketSize, EquihashType::NHashes);
        const uint32_t numBlocks = (endIdx - startIdx + ThreadsPerBlock - 1) / ThreadsPerBlock;

        const uint32_t collisionBitLength = EquihashType::CollisionBitLength;
        const uint32_t collisionBitMask = ((1U << collisionBitLength) - 1) << (round * collisionBitLength);

        const dim3 gridDim((numItems + ThreadsPerBlock - 1) / ThreadsPerBlock);
        const dim3 blockDim(ThreadsPerBlock);

        cudaKernel_detectCollisions<EquihashType><<<gridDim, blockDim>>>(
            hashes.get(), vCollisionPairs[bucketIdx].get() + collisionPairsOffset,
            vCollisionCounters[round][bucketIdx].get(), startIdx, endIdx, collisionBitMask);

        // Copy the collision count from device to host
        uint32_t collisionCount;
        copyToHost(&collisionCount, vCollisionCounters[round][bucketIdx].get(), sizeof(uint32_t));

        // Update the accumulated collision pair offset for the next bucket
        collisionPairsOffset += collisionCount;
    }

    // Store the accumulated collision pair offset for the current round
    m_vCollisionPairsOffsets[round] = collisionPairsOffset;
}

template<typename EquihashType>
__global__ void cudaKernel_xorCollisions(
    const uint32_t* hashes, uint32_t* xoredHashes,
    const uint32_t* const* collisionPairs, const uint32_t* const* collisionCounts,
    const uint32_t totalCollisionPairs, const uint32_t numBuckets)
{
    const uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid >= totalCollisionPairs)
        return;

    uint32_t collisionPairIdx = tid;
    uint32_t bucketIdx = 0;

    while (collisionPairIdx >= collisionCounts[bucketIdx][0] && bucketIdx < numBuckets)
    {
        collisionPairIdx -= collisionCounts[bucketIdx][0];
        ++bucketIdx;
    }

    const uint32_t collisionPair = collisionPairs[bucketIdx][collisionPairIdx];
    const uint32_t idx1 = collisionPair >> 16;
    const uint32_t idx2 = collisionPair & 0xFFFF;

    for (uint32_t i = 0; i < EquihashType::HashWords; ++i)
        xoredHashes[idx1 * EquihashType::HashWords + i] ^= hashes[idx2 * EquihashType::HashWords + i];
}

template<typename EquihashType>
void EhDevice<EquihashType>::xorCollisions()
{
    const uint32_t numBlocks = (vCollisionPairsOffsets[round] + ThreadsPerBlock - 1) / ThreadsPerBlock;

    const dim3 gridDim(numBlocks);
    const dim3 blockDim(ThreadsPerBlock);

    cudaKernel_xorCollisions<EquihashType><<<gridDim, blockDim>>>(
        hashes.get(), xoredHashes.get(), vCollisionPairs, 
        vCollisionCounters[round], vCollisionPairsOffsets[round], 
        EquihashType::NBucketCount);

    // Swap the hash pointers for the next round
    swap(hashes, xoredHashes);
}

/**
 * @brief Find valid solutions by checking the XORed values against the target difficulty.
 * 
 * @param hashes - Array of hash values 
 * @param solutions - Array to store the valid solutions
 * @param solutionCount - The number of valid solutions found 
 */
template<typename EquihashType>
__global__ void cudaKernel_findSolutions(
    const uint32_t* hashes,
    const uint32_t* const* collisionPairs, const uint32_t* const* collisionCounts,
    const uint32_t* collisionPairsOffsets, typename EquihashType::solution* solutions, uint32_t* solutionCount,
    const uint32_t numBuckets)
{
    const uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    const uint32_t totalCollisionPairs = collisionPairsOffsets[EquihashType::WK - 1];

    if (tid >= totalCollisionPairs)
        return;

    uint32_t collisionPairIdx = tid;
    uint32_t bucketIdx = 0;

    while (collisionPairIdx >= collisionCounts[EquihashType::WK - 1][bucketIdx] && bucketIdx < numBuckets)
    {
        collisionPairIdx -= collisionCounts[EquihashType::WK - 1][bucketIdx];
        ++bucketIdx;
    }

    const uint32_t collisionPair = collisionPairs[bucketIdx][collisionPairsOffsets[EquihashType::WK - 2] + collisionPairIdx];
    uint32_t indices[EquihashType::ProofSize] = { 0 };
    uint32_t xoredHash[EquihashType::HashWords] = { 0 };

    indices[0] = collisionPair >> 16;
    indices[1] = collisionPair & 0xFFFF;

    for (uint32_t i = 0; i < EquihashType::HashWords; ++i)
        xoredHash[i] = hashes[indices[0] * EquihashType::HashWords + i] ^ hashes[indices[1] * EquihashType::HashWords + i];

    uint32_t indicesCount = 2;

    for (uint32_t round = EquihashType::WK - 2; round >= 1; --round)
    {
        const uint32_t collisionPairsOffset = collisionPairsOffsets[round - 1];
        const uint32_t collisionPairsCount = collisionPairsOffsets[round] - collisionPairsOffset;

        bool found = false;

        for (uint32_t i = 0; i < collisionPairsCount; ++i)
        {
            const uint32_t pair = collisionPairs[bucketIdx][collisionPairsOffset + i];
            const uint32_t idx1 = pair >> 16;
            const uint32_t idx2 = pair & 0xFFFF;

            if (idx1 == indices[indicesCount - 2] || idx1 == indices[indicesCount - 1] ||
                idx2 == indices[indicesCount - 2] || idx2 == indices[indicesCount - 1])
            {
                const uint32_t newIndex = (idx1 == indices[indicesCount - 2] || idx1 == indices[indicesCount - 1]) ? idx2 : idx1;
                indices[indicesCount++] = newIndex;

                for (uint32_t j = 0; j < EquihashType::HashWords; ++j)
                    xoredHash[j] ^= hashes[newIndex * EquihashType::HashWords + j];

                found = true;
                break;
            }
        }

        if (!found)
            break;
    }

    if (indicesCount == EquihashType::ProofSize)
    {
        // Check if the xoredHash satisfies the difficulty target
        // TODO: Implement the difficulty check based on the specific target

        // If a valid solution is found, store it
        if (true/* Difficulty check passed */)
        {
            const uint32_t solutionIdx = atomicAdd(solutionCount, 1);
            for (uint32_t i = 0; i < EquihashType::ProofSize; ++i)
                solutions[solutionIdx].indices[i] = indices[i];
        }
    }
}

template<typename EquihashType>
uint32_t EhDevice<EquihashType>::findSolutions()
{
    const uint32_t numBlocks = (m_vCollisionPairsOffsets[round] + ThreadsPerBlock - 1) / ThreadsPerBlock;

    hipMemset(solutionCount.get(), 0, sizeof(uint32_t));

    const dim3 gridDim(numBlocks);
    const dim3 blockDim(ThreadsPerBlock);

    cudaKernel_findSolutions<EquihashType><<<gridDim, blockDim>>>(
        hashes.get(), vCollisionPairs, vCollisionCounters, m_vCollisionPairsOffsets, solutions.get(), solutionCount.get(), 
        EquihashType::NBucketCount);

    uint32_t nSolutionCount;
    copyToHost(&nSolutionCount, solutionCount.get(), sizeof(uint32_t));

    return nSolutionCount;
}

template<typename EquihashType>
void copySolutionsToHost(typename EquihashType::solution* devSolutions, const uint32_t nSolutionCount, vector<typename EquihashType::solution> &vHostSolutions)
{
    vHostSolutions.clear();
    // Resize the host solutions vector
    vHostSolutions.resize(nSolutionCount);

    // Copy the solutions from device to host
    copyToHost(vHostSolutions.data(), devSolutions, nSolutionCount * EquihashType::ProofSize);
}

// Explicit template instantiation
template void generateInitialHashes<Eh200_9>(
    const blake2b_state* devState, uint32_t* devHashes, const uint32_t threadsPerBlock);
template void detectCollisions<Eh200_9>(uint32_t* devHashes, uint32_t* devSlotBitmaps, const uint32_t threadsPerBlock);
template void xorCollisions<Eh200_9>(uint32_t* devHashes, uint32_t* devSlotBitmaps, uint32_t* devXoredHashes, const uint32_t threadsPerBlock);
template uint32_t findSolutions<Eh200_9>(uint32_t* devHashes, uint32_t* devSlotBitmaps, Eh200_9::solution* devSolutions, uint32_t* devSolutionCount, const uint32_t threadsPerBlock);
template void copySolutionsToHost<Eh200_9>(Eh200_9::solution* devSolutions, const uint32_t nSolutionCount, vector<Eh200_9::solution> &vHostSolutions);
