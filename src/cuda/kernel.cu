#include "hip/hip_runtime.h"

// Copyright (c) 2024 The Pastel developers
// Distributed under the MIT software license, see the accompanying
// file COPYING or https://www.opensource.org/licenses/mit-license.php.
#include <cstdint>
#include <vector>
#include <string>
#include <bitset>
#include <iostream>

#include <hip/hip_runtime.h>
#include <>

#include <src/cuda/kernel.h>
#include <src/cuda/memutils.h>
#include <src/cuda/blake2b_device.h>
#include <src/equihash/equihash.h>

using namespace std;

// Get the number of available CUDA devices
int getNumCudaDevices()
{
    int numDevices = 0;
    hipGetDeviceCount(&numDevices);
    return numDevices;
}

// Get the maximum number of threads per block supported by the device
int getMaxThreadsPerBlock(int deviceId)
{
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, deviceId);
    return deviceProp.maxThreadsPerBlock;
}

template<typename EquihashType>
bool EhDevice<EquihashType>::allocate_memory()
{
    try
    {
        // Allocate device memory for blake2b state
        initialState = make_cuda_unique<blake2b_state>(1);
        // Allocate device memory for initial hash values
        hashes = make_cuda_unique<uint32_t>(EquihashType::NHashWords);
        // Allocate device memory for XORed hash values
        xoredHashes = make_cuda_unique<uint32_t>(EquihashType::NHashWords);

        const uint32_t maxCollisionsPerBucket = 10000;

        // Allocate device buffer for collision pair pointers
        collisionPairs = make_cuda_unique<uint32_t*>(EquihashType::NBucketCount);
        vBucketCollisionPairs.resize(EquihashType::NBucketCount);
        for (uint32_t i = 0; i < EquihashType::NBucketCount; ++i)
        {
            vBucketCollisionPairs[i] = make_cuda_unique<uint32_t>(maxCollisionsPerBucket);
            collisionPairs.get()[i] = vBucketCollisionPairs[i].get();
        }

        collisionCounters = make_cuda_unique<uint32_t*>(EquihashType::WK);

        vCollisionCounters.resize(EquihashType::WK);
        for (uint32_t round = 0; round < EquihashType::WK; ++round)
        {
            vCollisionCounters[round].resize(EquihashType::NBucketCount);
            for (uint32_t i = 0; i < EquihashType::NBucketCount; ++i)
                vCollisionCounters[round][i] = make_cuda_unique<uint32_t>(1);
        }
        // Accumulated collision pair offsets for each round
        vCollisionPairsOffsets.resize(EquihashType::WK, 0);

        // Allocate device memory for solutions and solution count
        solutions = make_cuda_unique<typename EquihashType::solution>(MAXSOLUTIONS);
        solutionCount = make_cuda_unique<uint32_t>(1);

        return true;
    }
    catch (const std::exception& e)
    {
        std::cerr << "Memory allocation failed: " << e.what() << std::endl;
        return false;
    }
}

// Calculate the grid and block dimensions based on the problem size and device capabilities
void calculateGridAndBlockDims(dim3& gridDim, dim3& blockDim, size_t nHashes, int nThreadsPerHash, int deviceId)
{
    int nMaxThreadsPerBlock = getMaxThreadsPerBlock(deviceId);
    int nBlocks = (nHashes + nThreadsPerHash - 1) / nThreadsPerHash;

    blockDim.x = min(nThreadsPerHash, nMaxThreadsPerBlock);
    gridDim.x = (nBlocks + blockDim.x - 1) / blockDim.x;
}

// CUDA kernel to generate initial hashes from blake2b state
template<typename EquihashType>
__global__ void cudaKernel_generateInitialHashes(const blake2b_state* state, uint32_t* hashes)
{
    const uint32_t hashIdx = threadIdx.x + blockIdx.x * blockDim.x;

    if (hashIdx >= EquihashType::NHashes)
        return;

    const uint32_t blockIndex = hashIdx / EquihashType::IndicesPerHashOutput;

    blake2b_state localState = *state;
    blake2b_update_device(&localState, reinterpret_cast<const uint8_t*>(&blockIndex), sizeof(blockIndex));

    uint8_t hash[EquihashType::HashOutput];  
    blake2b_final_device(&localState, hash, EquihashType::HashOutput);

    const uint32_t outputIdx = hashIdx * EquihashType::HashWords;
    for (uint32_t i = 0; i < EquihashType::HashWords; ++i)
        hashes[outputIdx + i] = (reinterpret_cast<uint32_t*>(hash))[i];
}

template<typename EquihashType>
void EhDevice<EquihashType>::generateInitialHashes()
{
    const uint32_t numThreads = (EquihashType::NHashes + ThreadsPerBlock - 1) / ThreadsPerBlock * ThreadsPerBlock;

    dim3 gridDim((numThreads + ThreadsPerBlock - 1) / ThreadsPerBlock);
    dim3 blockDim(ThreadsPerBlock);

    cudaKernel_generateInitialHashes<EquihashType><<<gridDim, blockDim>>>(initialState.get(), hashes.get());
}

template <typename EquihashType>
__global__ void cudaKernel_detectCollisions(
    const uint32_t* hashes, uint32_t* collisionPairs, uint32_t* collisionCounts,
    const uint32_t startIdx, const uint32_t endIdx, const uint32_t collisionBitMask)
{
    const uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    const uint32_t idx = startIdx + tid;

    if (idx >= endIdx)
        return;

    const uint32_t hash = hashes[idx];
    const uint32_t maskedHash = hash & collisionBitMask;

    const uint32_t bucketIdx = idx / EquihashType::NBucketSize;
    const uint32_t bucketOffset = bucketIdx * EquihashType::NBucketSize;

    for (uint32_t i = bucketOffset; i < idx; ++i)
    {
        const uint32_t otherHash = hashes[i];
        const uint32_t otherMaskedHash = otherHash & collisionBitMask;

        if (maskedHash == otherMaskedHash)
        {
            const uint32_t collisionIdx = atomicAdd(collisionCounts, 1);
            collisionPairs[collisionIdx] = (idx << 16) | i;
        }
    }    
}

template <typename EquihashType>
void EhDevice<EquihashType>::detectCollisions()
{
    // Calculate the accumulated collision pair offset for the current round
    uint32_t collisionPairsOffset = 0;
    if (round > 0)
        collisionPairsOffset = vCollisionPairsOffsets[round - 1];

    const uint32_t collisionBitLength = EquihashType::CollisionBitLength;
    const uint32_t collisionBitMask = ((1U << collisionBitLength) - 1) << (round * collisionBitLength);

    // Set collision counter pointers to the corresponding device buffers for the current round
    for (uint32_t i = 0; i < EquihashType::NBucketCount; ++i)
        collisionCounters.get()[round * EquihashType::NBucketCount + i] = vCollisionCounters[round][i].get();

    for (uint32_t bucketIdx = 0; bucketIdx < EquihashType::NBucketCount; ++bucketIdx)
    {
        const uint32_t startIdx = bucketIdx * EquihashType::NBucketSize;
        const uint32_t endIdx = min(startIdx + EquihashType::NBucketSize, EquihashType::NHashes);
        const uint32_t numItems = endIdx - startIdx;

        const dim3 gridDim((numItems + ThreadsPerBlock - 1) / ThreadsPerBlock);
        const dim3 blockDim(ThreadsPerBlock);

        cudaKernel_detectCollisions<EquihashType><<<gridDim, blockDim>>>(
            hashes.get(), collisionPairs.get()[bucketIdx] + collisionPairsOffset,
            collisionCounters.get()[round * EquihashType::NBucketCount + bucketIdx], startIdx, endIdx, collisionBitMask);

        // Copy the collision count from device to host
        uint32_t collisionCount;
        copyToHost(&collisionCount, vCollisionCounters[round][bucketIdx].get(), sizeof(uint32_t));

        // Update the accumulated collision pair offset for the next bucket
        collisionPairsOffset += collisionCount;
    }

    // Store the accumulated collision pair offset for the current round
    vCollisionPairsOffsets[round] = collisionPairsOffset;
}

template<typename EquihashType>
__global__ void cudaKernel_xorCollisions(
    const uint32_t* hashes, uint32_t* xoredHashes,
    const uint32_t* const* collisionPairs, const uint32_t* const* collisionCounts,
    const uint32_t totalCollisionPairs)
{
    const uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid >= totalCollisionPairs)
        return;

    uint32_t collisionPairIdx = tid;
    uint32_t bucketIdx = 0;

    while (collisionPairIdx >= collisionCounts[bucketIdx][0] && bucketIdx < EquihashType::NBucketCount)
    {
        collisionPairIdx -= collisionCounts[bucketIdx][0];
        ++bucketIdx;
    }

    const uint32_t collisionPair = collisionPairs[bucketIdx][collisionPairIdx];
    const uint32_t idx1 = collisionPair >> 16;
    const uint32_t idx2 = collisionPair & 0xFFFF;

    for (uint32_t i = 0; i < EquihashType::HashWords; ++i)
        xoredHashes[idx1 * EquihashType::HashWords + i] ^= hashes[idx2 * EquihashType::HashWords + i];
}

template<typename EquihashType>
void EhDevice<EquihashType>::xorCollisions()
{
    const uint32_t numBlocks = (vCollisionPairsOffsets[round] + ThreadsPerBlock - 1) / ThreadsPerBlock;

    const dim3 gridDim(numBlocks);
    const dim3 blockDim(ThreadsPerBlock);

    cudaKernel_xorCollisions<EquihashType><<<gridDim, blockDim>>>(
        hashes.get(), xoredHashes.get(), collisionPairs.get(), 
        collisionCounters.get() + round * EquihashType::NBucketCount,
        vCollisionPairsOffsets[round]);

    // Swap the hash pointers for the next round
    swap(hashes, xoredHashes);
}

/**
 * @brief Find valid solutions by checking the XORed values against the target difficulty.
 * 
 * @param hashes - Array of hash values 
 * @param solutions - Array to store the valid solutions
 * @param solutionCount - The number of valid solutions found 
 */
template<typename EquihashType>
__global__ void cudaKernel_findSolutions(
    const uint32_t* hashes,
    const uint32_t* const* collisionPairs,
    const uint32_t* const* collisionCounts,
    const uint32_t* collisionPairsOffsets, typename EquihashType::solution* solutions, uint32_t* solutionCount)
{
    const uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    const uint32_t totalCollisionPairs = collisionPairsOffsets[EquihashType::WK - 1];

    if (tid >= totalCollisionPairs)
        return;

    uint32_t collisionPairIdx = tid;
    uint32_t bucketIdx = 0;

    while (collisionPairIdx >= collisionCounts[EquihashType::WK - 1][bucketIdx] && bucketIdx < EquihashType::NBucketCount)
    {
        collisionPairIdx -= collisionCounts[EquihashType::WK - 1][bucketIdx];
        ++bucketIdx;
    }

    const uint32_t collisionPair = collisionPairs[bucketIdx][collisionPairsOffsets[EquihashType::WK - 2] + collisionPairIdx];
    uint32_t indices[EquihashType::ProofSize] = { 0 };
    uint32_t xoredHash[EquihashType::HashWords] = { 0 };

    indices[0] = collisionPair >> 16;
    indices[1] = collisionPair & 0xFFFF;

    for (uint32_t i = 0; i < EquihashType::HashWords; ++i)
        xoredHash[i] = hashes[indices[0] * EquihashType::HashWords + i] ^ hashes[indices[1] * EquihashType::HashWords + i];

    uint32_t indicesCount = 2;

    for (uint32_t round = EquihashType::WK - 2; round >= 1; --round)
    {
        const uint32_t collisionPairsOffset = collisionPairsOffsets[round - 1];
        const uint32_t collisionPairsCount = collisionPairsOffsets[round] - collisionPairsOffset;

        bool found = false;

        for (uint32_t i = 0; i < collisionPairsCount; ++i)
        {
            const uint32_t pair = collisionPairs[bucketIdx][collisionPairsOffset + i];
            const uint32_t idx1 = pair >> 16;
            const uint32_t idx2 = pair & 0xFFFF;

            if (idx1 == indices[indicesCount - 2] || idx1 == indices[indicesCount - 1] ||
                idx2 == indices[indicesCount - 2] || idx2 == indices[indicesCount - 1])
            {
                const uint32_t newIndex = (idx1 == indices[indicesCount - 2] || idx1 == indices[indicesCount - 1]) ? idx2 : idx1;
                indices[indicesCount++] = newIndex;

                for (uint32_t j = 0; j < EquihashType::HashWords; ++j)
                    xoredHash[j] ^= hashes[newIndex * EquihashType::HashWords + j];

                found = true;
                break;
            }
        }

        if (!found)
            break;
    }

    if (indicesCount == EquihashType::ProofSize)
    {
        // Check if the xoredHash satisfies the difficulty target
        // TODO: Implement the difficulty check based on the specific target

        // If a valid solution is found, store it
        if (true/* Difficulty check passed */)
        {
            const uint32_t solutionIdx = atomicAdd(solutionCount, 1);
            for (uint32_t i = 0; i < EquihashType::ProofSize; ++i)
                solutions[solutionIdx].indices[i] = indices[i];
        }
    }
}

template<typename EquihashType>
uint32_t EhDevice<EquihashType>::findSolutions()
{
    const uint32_t numBlocks = (vCollisionPairsOffsets[round] + ThreadsPerBlock - 1) / ThreadsPerBlock;

    hipMemset(solutionCount.get(), 0, sizeof(uint32_t));

    const dim3 gridDim(numBlocks);
    const dim3 blockDim(ThreadsPerBlock);

    auto collisionPairsOffsets = make_cuda_unique<uint32_t>(vCollisionPairsOffsets.size());
    copyToDevice(collisionPairsOffsets.get(), vCollisionPairsOffsets.data(), vCollisionPairsOffsets.size() * sizeof(uint32_t));

    cudaKernel_findSolutions<EquihashType><<<gridDim, blockDim>>>(
        hashes.get(), collisionPairs.get(), collisionCounters.get(), collisionPairsOffsets.get(), solutions.get(), 
        solutionCount.get());

    uint32_t nSolutionCount;
    copyToHost(&nSolutionCount, solutionCount.get(), sizeof(uint32_t));

    return nSolutionCount;
}

template<typename EquihashType>
void EhDevice<EquihashType>::debugPrintHashes()
{
    v_uint32 hostHashes(EquihashType::NHashWords);
    copyToHost(hostHashes.data(), hashes.get(), hostHashes.size() * sizeof(uint32_t));

    // Print out the generated hashes
    size_t hashNo = 0;
    for (size_t i = 0; i < hostHashes.size(); i += EquihashType::HashWords)
    {
        ++hashNo;
        if (hashNo % 0x1000 != 0)
            continue;
        cout << "Hash " << dec << hashNo << ": ";
        bool bAllZeroes = true;
        for (size_t j = 0; j < EquihashType::HashWords; ++j)
        {
            if (hostHashes[i + j])
                bAllZeroes = false;
            cout << hex << hostHashes[i + j] << " ";
        }
        if (bAllZeroes)
        {
            cout << "All zeroes" << endl;
            break;
        }
        cout << endl;
    }
}

template<typename EquihashType>
uint32_t EhDevice<EquihashType>::solver()
{
    // Generate initial hash values
    generateInitialHashes();

    // Perform K rounds of collision detection and XORing
    for (uint32_t round = 0; round < EquihashType::WK; round++)
    {
        // Detect collisions and XOR the colliding pairs
        detectCollisions();
        xorCollisions();
    }

    return findSolutions();
}

template<typename EquihashType>
void EhDevice<EquihashType>::copySolutionsToHost(vector<typename EquihashType::solution> &vHostSolutions)
{
    uint32_t nSolutionCount = 0;
    copyToHost(&nSolutionCount, solutionCount.get(), sizeof(uint32_t));

    vHostSolutions.clear();
    // Resize the host solutions vector
    vHostSolutions.resize(nSolutionCount);

    // Copy the solutions from device to host
    copyToHost(vHostSolutions.data(), solutions.get(), nSolutionCount * EquihashType::ProofSize);
}

// Explicit template instantiation
template class EhDevice<Eh200_9>;
