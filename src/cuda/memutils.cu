// Copyright (c) 2024 The Pastel developers
// Distributed under the MIT software license, see the accompanying
// file COPYING or https://www.opensource.org/licenses/mit-license.php.
#include <iostream>

#include <src/cuda/memutils.h>
#include <src/equihash/equihash.h>
#include <blake2b.h>

using namespace std;

#define CUDA_CHECK(call)                                                   \
do {                                                                       \
    hipError_t err = call;                                                \
    if (err != hipSuccess) {                                              \
        std::cerr << "CUDA error: " << hipGetErrorString(err) << " in "   \
                  << __FILE__ << ":" << __LINE__ << std::endl;             \
        exit(EXIT_FAILURE);                                                \
    }                                                                      \
} while (0)

// Allocate device memory
void allocateDeviceMemory(void** devPtr, size_t size)
{
    CUDA_CHECK(hipMalloc(devPtr, size));
}

// Free device memory
void freeDeviceMemory(void* devPtr)
{
    CUDA_CHECK(hipFree(devPtr));
}

// Copy data from host to device
void copyToDevice(void* dst, const void* src, const size_t size)
{
    CUDA_CHECK(hipMemcpy(dst, src, size, hipMemcpyHostToDevice));
}

// Copy data from device to host
void copyToHost(void* dst, const void* src, const size_t size)
{
    CUDA_CHECK(hipMemcpy(dst, src, size, hipMemcpyDeviceToHost));
}

void CudaDeleter::operator()(void* ptr) const
{
    hipFree(ptr);
}

// Helper function to create a unique_ptr with CUDA memory
template <typename T>
unique_ptr<T, CudaDeleter> make_cuda_unique(const size_t numElements)
{
    T* devPtr = nullptr;
    CUDA_CHECK(hipMalloc(&devPtr, numElements * sizeof(T)));
    return unique_ptr<T, CudaDeleter>(devPtr);
}

template <typename T>
unique_ptr<T, CudaDeleter> make_cuda_unique_2d(const size_t rowSize, const size_t columnSize, size_t &pitch)
{
    T* devPtr = nullptr;
    CUDA_CHECK(hipMallocPitch(&devPtr, &pitch, rowSize * sizeof(T), columnSize));
    return unique_ptr<T, CudaDeleter>(devPtr);
}

template std::unique_ptr<uint32_t, CudaDeleter> make_cuda_unique<uint32_t>(const size_t numElements);
template std::unique_ptr<uint32_t*, CudaDeleter> make_cuda_unique<uint32_t*>(const size_t numElements);
template std::unique_ptr<blake2b_state, CudaDeleter> make_cuda_unique<blake2b_state>(const size_t numElements);
template std::unique_ptr<Eh200_9::solution_device_type, CudaDeleter> make_cuda_unique<Eh200_9::solution_device_type>(const size_t numElements);

template std::unique_ptr<uint32_t, CudaDeleter> make_cuda_unique_2d(const size_t rowSize, const size_t columnSize, size_t &pitch);